/*
 * Copyright 2019 Saman Ashkiani
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 * http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or
 * implied. See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <stdio.h>
#include <time.h>
#include <unistd.h>
#include <algorithm>
#include <cstdlib>
#include <iostream>
#include <random>
#include <thread>
#include <vector>
#include "coordinate_hash_map.cuh"

//=======================================
#define DEVICE_ID 0

int main(int argc, char** argv) {
    //=========
    int devCount;
    hipGetDeviceCount(&devCount);
    hipDeviceProp_t devProp;
    if (devCount) {
        hipSetDevice(DEVICE_ID);  // be changed later
        hipGetDeviceProperties(&devProp, DEVICE_ID);
    }
    printf("Device: %s\n", devProp.name);

    /******** Hash table meta data ********/
    uint32_t num_elems = 1 << 11;
    float expected_chain = 0.8f;
    uint32_t num_elems_per_bucket = 15;
    uint32_t expected_elems_per_bucket = expected_chain * num_elems_per_bucket;
    uint32_t num_buckets = (num_elems + expected_elems_per_bucket - 1) /
                           expected_elems_per_bucket;

    /******** Insertion data ********/
    using KeyT = uint64_t;
    constexpr size_t D = 7;
    using ValueT = uint32_t;
    using HashFunc = CoordinateHashFunc<KeyT, D>;
    using KeyTD = Coordinate<KeyT, D>;

    const int num_insertions = num_elems / 2;
    std::vector<KeyTD> h_key(num_insertions);
    std::vector<ValueT> h_value(num_insertions);
    std::vector<ValueT> h_result_gt(num_insertions);
    std::vector<uint8_t> h_found_gt(num_insertions);
    const int64_t seed = 1;
    std::mt19937 rng(seed);
    std::vector<uint32_t> index(num_insertions * D);
    std::iota(index.begin(), index.end(), 0);
    std::shuffle(index.begin(), index.end(), rng);

    for (int32_t i = 0; i < num_insertions; ++i) {
      for (int d = 0; d < D; ++d) {
        h_key[i][d] = index[i * D + d];
      }
      h_result_gt[i] = h_value[i] = i;
      h_found_gt[i] = 1;
    }

    /******* Instantiate hash table ********/
    printf("num elems: %d, num buckets: %d -- num insertions: %d\n", num_elems,
           num_buckets, num_insertions);
    CoordinateHashMap<KeyT, D, ValueT, HashFunc> hash_table(num_elems);

    /****** Insert and query ********/
    float build_time = 0;
    hash_table.Insert(h_key, h_value, build_time);
    printf("1) Insert finished in %.3f ms (%.3f M elements/s)\n", build_time,
           double(num_insertions) / (build_time * 1000.0));

    std::vector<ValueT> h_result(num_insertions);
    std::vector<uint8_t> h_found(num_insertions);
    float search_time = 0;
    hash_table.Search(h_key, h_result, h_found, search_time);
    printf("2) Query finished in %.3f ms (%.3f M queries/s)\n", search_time,
           double(num_insertions) / (search_time * 1000.0));

    bool search_success = true;
    for (int i = 0; i < num_insertions; i++) {
        if (!h_found_gt[i] && h_found[i]) {
            printf("### wrong result at index %d: should be NOT FOUND\n", i);
            search_success = false;
        }
        if (h_found_gt[i] && !h_found[i]) {
            printf("### wrong result at index %d: should be FOUND\n", i);
            search_success = false;
        }
        if (h_found_gt[i] && h_found[i] && (h_result_gt[i] != h_result[i])) {
            printf("### wrong result at index %d: [%d] -> %d, but should be "
                   "%d\n",
                   i, h_key[i][0], h_result[i], h_result_gt[i]);
            search_success = false;
        }
    }
    if (search_success) {
        printf("2) Validation done\n");
    }
    double load_factor = hash_table.ComputeLoadFactor(1);
    printf("The load factor is %.2f, number of buckets %d\n", load_factor,
           num_buckets);

    /** Disturb the value **/
    for (auto& v : h_value) {
        v += 1;
    }

    std::this_thread::sleep_for(std::chrono::seconds(2));

    /** Insert again **/
    hash_table.Insert(h_key, h_value, build_time);
    printf("3) Insert finished in %.3f ms (%.3f M elements/s)\n", build_time,
           double(num_insertions) / (build_time * 1000.0));

    hash_table.Search(h_key, h_result, h_found, search_time);
    printf("4) Query finished in %.3f ms (%.3f M queries/s)\n", search_time,
           double(num_insertions) / (search_time * 1000.0));

    search_success = true;
    for (int i = 0; i < num_insertions; i++) {
        if (!h_found_gt[i] && h_found[i]) {
            printf("### wrong result at index %d: should be NOT FOUND\n", i);
            search_success = false;
        }
        if (h_found_gt[i] && !h_found[i]) {
            printf("### wrong result at index %d: should be FOUND\n", i);
            search_success = false;
        }
        if (h_found_gt[i] && h_found[i] && (h_result_gt[i] != h_result[i])) {
            printf("### wrong result at index %d: [%d] -> %d, but should be "
                   "%d\n",
                   i, h_key[i][0], h_result[i], h_result_gt[i]);
            search_success = false;
        }
    }
    if (search_success) {
        printf("4) Validation done\n");
    }

    load_factor = hash_table.ComputeLoadFactor(1);
    printf("The load factor is %.2f, number of buckets %d\n", load_factor,
           num_buckets);

    return 0;
}